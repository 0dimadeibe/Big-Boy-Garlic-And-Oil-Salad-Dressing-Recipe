#include "hip/hip_runtime.h"
// #include <algorithm>
#include <hip/hip_vector_types.h>
#include <limits>
#include <memory>
#include <optional>
#include <string_view>

#include "hip/hip_runtime.h"

using namespace std::literals;

#define SUNSHINE_STRINGVIEW_HELPER(x) x##sv
#define SUNSHINE_STRINGVIEW(x) SUNSHINE_STRINGVIEW_HELPER(x)

#define CU_CHECK(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return -1

#define CU_CHECK_VOID(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return;

#define CU_CHECK_PTR(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return nullptr;

#define CU_CHECK_IGNORE(x, y) \
  check((x), SUNSHINE_STRINGVIEW(y ": "))

using namespace std::literals;

//////////////////// Special desclarations
/**
 * NVCC segfaults when including <chrono>
 * Therefore, some declarations need to be added explicitely
 */
namespace platf {
struct img_t {
public:
  std::uint8_t *data {};
  std::int32_t width {};
  std::int32_t height {};
  std::int32_t pixel_pitch {};
  std::int32_t row_pitch {};

  virtual ~img_t() = default;
};
} // namespace platf

namespace video {
using __float4 = float[4];
using __float3 = float[3];
using __float2 = float[2];

struct __attribute__((__aligned__(16))) color_t {
  float4 color_vec_y;
  float4 color_vec_u;
  float4 color_vec_v;
  float2 range_y;
  float2 range_uv;
};

struct __attribute__((__aligned__(16))) color_extern_t {
  __float4 color_vec_y;
  __float4 color_vec_u;
  __float4 color_vec_v;
  __float2 range_y;
  __float2 range_uv;
};

extern color_extern_t colors[4];
} // namespace video

//////////////////// End special declarations

namespace cuda {
auto constexpr INVALID_TEXTURE = std::numeric_limits<hipTextureObject_t>::max();

template<class T>
inline T div_align(T l, T r) {
  return (l + r - 1) / r;
}

void pass_error(const std::string_view &sv, const char *name, const char *description);
inline static int check(hipError_t result, const std::string_view &sv) {
  if(result) {
    auto name        = hipGetErrorName(result);
    auto description = hipGetErrorString(result);

    pass_error(sv, name, description);
    return -1;
  }

  return 0;
}

__device__ __constant__ video::color_t color;


inline __device__ float3 bgra_to_rgb(uchar4 vec) {
  return make_float3((float)vec.z, (float)vec.y, (float)vec.x);
}

inline __device__ float2 calcUV(float3 pixel) {
  float4 vec_u = color.color_vec_u;
  float4 vec_v = color.color_vec_v;

  float u = dot(pixel, make_float3(vec_u)) + vec_u.w;
  float v = dot(pixel, make_float3(vec_v)) + vec_v.w;

  u = u * color.range_uv.x + color.range_uv.y;
  v = (v * color.range_uv.x + color.range_uv.y) * 224.0f / 256.0f + 0.0625f * 256.0f;

  return make_float2(u, v);
}

inline __device__ float calcY(float3 pixel) {
  float4 vec_y = color.color_vec_y;

  return (dot(pixel, make_float3(vec_y)) + vec_y.w) * color.range_y.x + color.range_y.y;
}

__global__ void RGBA_to_NV12(
  hipTextureObject_t srcImage, std::uint8_t *dstY, std::uint8_t *dstUV,
  std::uint32_t dstPitchY, std::uint32_t dstPitchUV,
  std::uint32_t width, std::uint32_t height) {

  int idX = (threadIdx.x + blockDim.x * blockIdx.x) * 2;
  int idY = (threadIdx.y + blockDim.y * blockIdx.y);

  if(idX >= width) return;
  if(idY >= height) return;

  dstY  = dstY + idX + idY * dstPitchY;
  dstUV = dstUV + idX + (idY / 2 * dstPitchUV);

  float x = (float)idX / (float)width / 4;
  float y = (float)idY / (float)height;

  float3 rgb_l = bgra_to_rgb(tex2D<uchar4>(srcImage, x, y));
  float3 rgb_r = bgra_to_rgb(tex2D<uchar4>(srcImage, x + 0.25f / width, y + 1.0f / height));

  float2 uv = calcUV((rgb_l + rgb_r) * 0.5f);

  dstUV[0] = uv.x;
  dstUV[1] = uv.y;
  dstY[0]  = calcY(rgb_l);
  dstY[1]  = calcY(rgb_r);
}

sws_t::sws_t(int in_width, int in_height, int out_width, int out_height, int threadsPerBlock)
    : array {}, texture { INVALID_TEXTURE }, width { out_width }, height { out_height }, threadsPerBlock { threadsPerBlock } {
  auto format = hipCreateChannelDesc<uchar4>();

  CU_CHECK_VOID(hipMallocArray(&array, &format, in_width, in_height, hipArrayDefault), "Couldn't allocate cuda array");

  hipResourceDesc res {};
  res.resType         = hipResourceTypeArray;
  res.res.array.array = array;

  hipTextureDesc desc {};

  desc.readMode         = hipReadModeElementType;
  desc.filterMode       = hipFilterModePoint;
  desc.normalizedCoords = true;

  std::fill_n(std::begin(desc.addressMode), 2, hipAddressModeClamp);

  CU_CHECK_VOID(hipCreateTextureObject(&texture, &res, &desc, nullptr), "Couldn't create cuda texture");
}

sws_t::~sws_t() {
  if(texture != INVALID_TEXTURE) {
    CU_CHECK_IGNORE(hipDestroyTextureObject(texture), "Couldn't deallocate cuda texture");

    texture = INVALID_TEXTURE;
  }

  if(array) {
    CU_CHECK_IGNORE(hipFreeArray(array), "Couldn't deallocate cuda array");

    array = hipArray_t {};
  }
}

std::unique_ptr<sws_t> sws_t::make(int in_width, int in_height, int out_width, int out_height) {
  hipDeviceProp_t props;
  int device;
  CU_CHECK_PTR(hipGetDevice(&device), "Couldn't get cuda device");
  CU_CHECK_PTR(hipGetDeviceProperties(&props, device), "Couldn't get cuda device properties");

  auto sws = std::make_unique<sws_t>(in_width, in_height, out_width, out_height, props.maxThreadsPerMultiProcessor / props.maxBlocksPerMultiProcessor / 2);

  if(sws->texture == INVALID_TEXTURE) {
    return nullptr;
  }

  return sws;
}

int sws_t::convert(std::uint8_t *Y, std::uint8_t *UV, std::uint32_t pitchY, std::uint32_t pitchUV) {
  int threadsX = width / 2;
  int threadsY = height;

  dim3 block(threadsPerBlock, threadsPerBlock);
  dim3 grid(div_align(threadsX, threadsPerBlock), div_align(threadsY, threadsPerBlock));

  RGBA_to_NV12<<<block, grid>>>(texture, Y, UV, pitchY, pitchUV, width, height);

  return CU_CHECK_IGNORE(hipGetLastError(), "RGBA_to_NV12 failed");
}

void sws_t::set_colorspace(std::uint32_t colorspace, std::uint32_t color_range) {
  color_range = 1;
  colorspace = 5;
  video::color_extern_t *color_p;
  switch(colorspace) {
  case 5: // SWS_CS_SMPTE170M
    color_p = &video::colors[0];
    break;
  case 1: // SWS_CS_ITU709
    color_p = &video::colors[2];
    break;
  case 9: // SWS_CS_BT2020
  default:
    color_p = &video::colors[0];
  };

  if(color_range > 1) {
    // Full range
    ++color_p;
  }

  auto color_matrix = *(video::color_t*)color_p;
  color_matrix.color_vec_y.w *= 256.0f;
  color_matrix.color_vec_u.w *= 256.0f;
  color_matrix.color_vec_v.w *= 256.0f;

  color_matrix.range_y.y *= 256.0f;
  color_matrix.range_uv.y *= 256.0f;

  static_assert(sizeof(video::color_t) == sizeof(video::color_extern_t), "color matrix struct mismatch");

  CU_CHECK_IGNORE(hipMemcpyToSymbol(HIP_SYMBOL(color), &color_matrix, sizeof(video::color_t)), "Couldn't copy color matrix to cuda");
}

int sws_t::load_ram(platf::img_t &img) {
  return CU_CHECK_IGNORE(hipMemcpy2DToArray(array, 0, 0, img.data, img.row_pitch, img.width * img.pixel_pitch, img.height, hipMemcpyHostToDevice), "Couldn't copy to cuda array");
}

} // namespace cuda