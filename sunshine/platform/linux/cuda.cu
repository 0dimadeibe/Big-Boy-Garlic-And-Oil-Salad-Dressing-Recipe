#include "hip/hip_runtime.h"
// #include <algorithm>
#include <hip/hip_vector_types.h>
#include <limits>
#include <memory>
#include <optional>
#include <string_view>

#include "hip/hip_runtime.h"

using namespace std::literals;

#define SUNSHINE_STRINGVIEW_HELPER(x) x##sv
#define SUNSHINE_STRINGVIEW(x) SUNSHINE_STRINGVIEW_HELPER(x)

#define CU_CHECK(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return -1

#define CU_CHECK_VOID(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return;

#define CU_CHECK_PTR(x, y) \
  if(check((x), SUNSHINE_STRINGVIEW(y ": "))) return nullptr;

#define CU_CHECK_IGNORE(x, y) \
  check((x), SUNSHINE_STRINGVIEW(y ": "))

using namespace std::literals;

//////////////////// Special desclarations
/**
 * NVCC segfaults when including <chrono>
 * Therefore, some declarations need to be added explicitely
 */
namespace platf {
struct img_t {
public:
  std::uint8_t *data {};
  std::int32_t width {};
  std::int32_t height {};
  std::int32_t pixel_pitch {};
  std::int32_t row_pitch {};

  virtual ~img_t() = default;
};
} // namespace platf

namespace video {
using __float4 = float[4];
using __float3 = float[3];
using __float2 = float[2];

struct __attribute__((__aligned__(16))) color_t {
  float4 color_vec_y;
  float4 color_vec_u;
  float4 color_vec_v;
  float2 range_y;
  float2 range_uv;
};

struct __attribute__((__aligned__(16))) color_extern_t {
  __float4 color_vec_y;
  __float4 color_vec_u;
  __float4 color_vec_v;
  __float2 range_y;
  __float2 range_uv;
};

static_assert(sizeof(video::color_t) == sizeof(video::color_extern_t), "color matrix struct mismatch");

extern color_t colors[4];
} // namespace video

//////////////////// End special declarations

namespace cuda {
auto constexpr INVALID_TEXTURE = std::numeric_limits<hipTextureObject_t>::max();

template<class T>
inline T div_align(T l, T r) {
  return (l + r - 1) / r;
}

void pass_error(const std::string_view &sv, const char *name, const char *description);
inline static int check(hipError_t result, const std::string_view &sv) {
  if(result) {
    auto name        = hipGetErrorName(result);
    auto description = hipGetErrorString(result);

    pass_error(sv, name, description);
    return -1;
  }

  return 0;
}

template<class T>
ptr_t make_ptr() {
  void *p;
  CU_CHECK_PTR(hipMalloc(&p, sizeof(T)), "Couldn't allocate color matrix");

  ptr_t ptr { p };

  return ptr;
}

void freeCudaPtr_t::operator()(void *ptr) {
  CU_CHECK_IGNORE(hipFree(ptr), "Couldn't free cuda device pointer");
}

inline __device__ float3 bgra_to_rgb(uchar4 vec) {
  return make_float3((float)vec.z, (float)vec.y, (float)vec.x);
}

inline __device__ float2 calcUV(float3 pixel, const video::color_t *const color_matrix) {
  float4 vec_u = color_matrix->color_vec_u;
  float4 vec_v = color_matrix->color_vec_v;

  float u = dot(pixel, make_float3(vec_u)) + vec_u.w;
  float v = dot(pixel, make_float3(vec_v)) + vec_v.w;

  u = u * color_matrix->range_uv.x + color_matrix->range_uv.y;
  v = (v * color_matrix->range_uv.x + color_matrix->range_uv.y) * 224.0f / 256.0f + 0.0625f * 256.0f;

  return make_float2(u, v);
}

inline __device__ float calcY(float3 pixel, const video::color_t *const color_matrix) {
  float4 vec_y = color_matrix->color_vec_y;

  return (dot(pixel, make_float3(vec_y)) + vec_y.w) * color_matrix->range_y.x + color_matrix->range_y.y;
}

__global__ void RGBA_to_NV12(
  hipTextureObject_t srcImage, std::uint8_t *dstY, std::uint8_t *dstUV,
  std::uint32_t dstPitchY, std::uint32_t dstPitchUV,
  const viewport_t viewport, const video::color_t *const color_matrix) {

  int idX = (threadIdx.x + blockDim.x * blockIdx.x) * 2;
  int idY = (threadIdx.y + blockDim.y * blockIdx.y);

  if(idX >= viewport.width) return;
  if(idY >= viewport.height) return;

  float x = (float)idX / (float)viewport.width / 4;
  float y = (float)idY / (float)viewport.height;

  idX += viewport.offsetX;
  idY += viewport.offsetY;

  dstY  = dstY + idX + idY * dstPitchY;
  dstUV = dstUV + idX + (idY / 2 * dstPitchUV);

  float3 rgb_l = bgra_to_rgb(tex2D<uchar4>(srcImage, x, y));
  float3 rgb_r = bgra_to_rgb(tex2D<uchar4>(srcImage, x + 0.25f / viewport.width, y + 1.0f / viewport.height));

  float2 uv = calcUV((rgb_l + rgb_r) * 0.5f, color_matrix);

  dstUV[0] = uv.x;
  dstUV[1] = uv.y;
  dstY[0]  = calcY(rgb_l, color_matrix);
  dstY[1]  = calcY(rgb_r, color_matrix);
}

sws_t::sws_t(int in_width, int in_height, int out_width, int out_height, int pitch, int threadsPerBlock, ptr_t &&color_matrix)
    : array {}, texture { INVALID_TEXTURE }, threadsPerBlock { threadsPerBlock }, color_matrix { std::move(color_matrix) } {
  auto format = hipCreateChannelDesc<uchar4>();

  CU_CHECK_VOID(hipMallocArray(&array, &format, pitch, in_height, hipArrayDefault), "Couldn't allocate cuda array");

  hipResourceDesc res {};
  res.resType         = hipResourceTypeArray;
  res.res.array.array = array;

  hipTextureDesc desc {};

  desc.readMode         = hipReadModeElementType;
  desc.filterMode       = hipFilterModePoint;
  desc.normalizedCoords = true;

  std::fill_n(std::begin(desc.addressMode), 2, hipAddressModeClamp);

  CU_CHECK_VOID(hipCreateTextureObject(&texture, &res, &desc, nullptr), "Couldn't create cuda texture");


  // Ensure aspect ratio is maintained
  auto scalar       = std::fminf(out_width / (float)in_width, out_height / (float)in_height);
  auto out_width_f  = in_width * scalar;
  auto out_height_f = in_height * scalar;

  // result is always positive
  auto offsetX_f = (out_width - out_width_f) / 2;
  auto offsetY_f = (out_height - out_height_f) / 2;

  viewport.width  = out_width_f;
  viewport.height = out_height_f;

  viewport.offsetX = offsetX_f;
  viewport.offsetY = offsetY_f;
}

sws_t::~sws_t() {
  if(texture != INVALID_TEXTURE) {
    CU_CHECK_IGNORE(hipDestroyTextureObject(texture), "Couldn't deallocate cuda texture");

    texture = INVALID_TEXTURE;
  }

  if(array) {
    CU_CHECK_IGNORE(hipFreeArray(array), "Couldn't deallocate cuda array");

    array = hipArray_t {};
  }
}

std::unique_ptr<sws_t> sws_t::make(int in_width, int in_height, int out_width, int out_height, int pitch) {
  hipDeviceProp_t props;
  int device;
  CU_CHECK_PTR(hipGetDevice(&device), "Couldn't get cuda device");
  CU_CHECK_PTR(hipGetDeviceProperties(&props, device), "Couldn't get cuda device properties");

  auto ptr = make_ptr<video::color_t>();
  if(!ptr) {
    return nullptr;
  }

  auto sws = std::make_unique<sws_t>(in_width, in_height, out_width, out_height, pitch, props.maxThreadsPerMultiProcessor / props.maxBlocksPerMultiProcessor / 2, std::move(ptr));

  if(sws->texture == INVALID_TEXTURE) {
    return nullptr;
  }

  return sws;
}

int sws_t::convert(std::uint8_t *Y, std::uint8_t *UV, std::uint32_t pitchY, std::uint32_t pitchUV) {
  return convert(Y, UV, pitchY, pitchUV, viewport);
}

int sws_t::convert(std::uint8_t *Y, std::uint8_t *UV, std::uint32_t pitchY, std::uint32_t pitchUV, const viewport_t &viewport) {
  int threadsX = viewport.width / 2;
  int threadsY = viewport.height;

  dim3 block(threadsPerBlock, threadsPerBlock);
  dim3 grid(div_align(threadsX, threadsPerBlock), div_align(threadsY, threadsPerBlock));

  RGBA_to_NV12<<<block, grid>>>(texture, Y, UV, pitchY, pitchUV, viewport, (video::color_t*)color_matrix.get());

  return CU_CHECK_IGNORE(hipGetLastError(), "RGBA_to_NV12 failed");
}

void sws_t::set_colorspace(std::uint32_t colorspace, std::uint32_t color_range) {
  video::color_t *color_p;
  switch(colorspace) {
  case 5: // SWS_CS_SMPTE170M
    color_p = &video::colors[0];
    break;
  case 1: // SWS_CS_ITU709
    color_p = &video::colors[2];
    break;
  case 9: // SWS_CS_BT2020
  default:
    color_p = &video::colors[0];
  };

  if(color_range > 1) {
    // Full range
    ++color_p;
  }

  auto color_matrix = *color_p;
  color_matrix.color_vec_y.w *= 256.0f;
  color_matrix.color_vec_u.w *= 256.0f;
  color_matrix.color_vec_v.w *= 256.0f;

  color_matrix.range_y.y *= 256.0f;
  color_matrix.range_uv.y *= 256.0f;

  CU_CHECK_IGNORE(hipMemcpy(this->color_matrix.get(), &color_matrix, sizeof(video::color_t), hipMemcpyHostToDevice), "Couldn't copy color matrix to cuda");
}

int sws_t::load_ram(platf::img_t &img) {
  return CU_CHECK_IGNORE(hipMemcpy2DToArray(array, 0, 0, img.data, img.row_pitch, img.width * img.pixel_pitch, img.height, hipMemcpyHostToDevice), "Couldn't copy to cuda array");
}

} // namespace cuda